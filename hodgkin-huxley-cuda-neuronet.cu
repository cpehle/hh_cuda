#include "hip/hip_runtime.h"
#include <cstdio>
#include <fstream>
#include <sstream>
#include <cmath>
#include <cstdlib>
#include <climits>
#include <ctime>
#include "hodgkin-huxley-cuda-neuronet.h"

unsigned int seed = 1;
float h = 0.1f;
float SimulationTime = 10000.0f; // in ms

int Nneur = 20;
int W_P_NUM_BUND = 1; // number of different poisson weights
int W_P_BUND_SZ = Nneur/W_P_NUM_BUND; // Number of neurons in bundle with same w_ps
int BUND_SZ = 2;  // Number of neurons in a single realization
int NUM_BUND = W_P_BUND_SZ/BUND_SZ;

// connection parameters
float I_e = 5.27f;
float w_p_start = 1.8f; // pA
float w_p_stop = 2.0f;
float w_n = 5.4f;
float rate = 200.0f;

char f_name[500] = "0";
char par_f_name[500] = "0";

__device__ __host__ float get_random(unsigned int *seed){
	// return random number homogeneously distributed in interval [0:1]
	unsigned long a = 16807;
	unsigned long m = 2147483647;
	unsigned long x = (unsigned long) *seed;
	x = (a * x) % m;
	*seed = (unsigned int) x;
	return ((float)x)/m;
}

__device__ float hh_Vm(float V, float n_ch, float m_ch, float h_ch, float I_syn, float I_e, float h){
	return (I_e - g_K*(V - E_K)*n_ch*n_ch*n_ch*n_ch - g_Na*(V - E_Na)*m_ch*m_ch*m_ch*h_ch - g_L*(V - E_L) + I_syn)*h*Cm;
}

__device__ float hh_n_ch(float V, float n_ch, float h){
	float temp = 1.0f - expf(-(V + 55.0f)*0.1f);
	if (temp != 0.0f){
		return (.01f*(1.0f - n_ch)*(V + 55.0f)/temp - 0.125*n_ch*expf(-(V + 65.0f)*0.0125f))*h;
	} else {
//		printf("Деление на ноль, n! \n");
//      For understanding why what, calculate the limit for v/(1 - exp(v/10)) then v tend to 0
		return (0.1f*(1.0f - n_ch)- 0.125*n_ch*expf(-(V + 65.0f)*0.0125f))*h;
	}
}

__device__ float hh_m_ch(float V, float m_ch, float h){
	float temp = 1.0f - expf(-(V + 40.0f)*0.1f);
	if (temp != 0.0f){
		return (0.1f*(1.0f - m_ch)*(V + 40.0f)/temp - 4.0f*m_ch*expf(-(V + 65.0f)*0.055555556f))*h;
	} else {
//		printf("Деление на ноль, m! \n");
		return ((1.0f - m_ch) - 4.0f*m_ch*expf(-(V + 65.0f)*0.055555556f))*h;
	}
}

__device__ float hh_h_ch(float V, float h_ch, float h){
	return (.07f*(1.0f - h_ch)*expf(-(V + 65.0f)*0.05f) - h_ch/(1.0f + expf(-(V + 35.0f)*0.1f)))*h;
}

__global__ void init_poisson(int* psn_time, unsigned int *psn_seed, unsigned int seed, float rate, float h, int Nneur, int BundleSize){
	int n = blockIdx.x*blockDim.x + threadIdx.x;
	int neur = n % BundleSize;
	if (n < Nneur){
		psn_seed[n] = seed + 100000*(neur + 1);
		psn_time[n] = -(1000.0f/(h*rate))*logf(get_random(psn_seed + n));
	}
}

__global__ void integrate_synapses(float* y, float* weight, int* delay, int* pre_conn, int* post_conn,
		int* spike_time, int* num_spike_syn, int* num_spike_neur, int t, int Nneur, int Ncon){
	int s = blockDim.x*blockIdx.x + threadIdx.x;
	if (s < Ncon){
		int pre_neur = pre_conn[s];
		// if we processed less spikes than there is in presynaptic neuron
		// we need to check whether new spikes at arrive this moment of time
		if (num_spike_syn[s] < num_spike_neur[pre_neur]){
			if (spike_time[Nneur*num_spike_syn[s] + pre_neur] == t - delay[s]){
				atomicAdd(&y[post_conn[s]], weight[s]);
				num_spike_syn[s]++;
			}
		}
	}
}

__global__ void integrate_neurons(
		float* V_m, float* V_m_last, float* n_ch, float* m_ch, float* h_ch,
		int* spike_time, int* num_spike_neur,
		float* I_e, float* y, float* I_syn, float* y_psn, float* I_psn, int* psn_time, unsigned int* psn_seed,
		float* I_syn_last, float* exp_w_p, float exp_psc, float rate,
		int Nneur, int t, float h){
		int n = blockIdx.x*blockDim.x + threadIdx.x;
		if (n < Nneur){
			I_psn[n]  = (y_psn[n]*h + I_psn[n])*exp_psc;
			y_psn[n] *= exp_psc;

			I_syn[n]  = (y[n]*h + I_syn[n])*exp_psc;
			y[n] *= exp_psc;

			// if where is poisson impulse on neuron
			while (psn_time[n] == t){
				y_psn[n] += exp_w_p[n];
				psn_time[n] -= (1000.0f/(rate*h))*logf(get_random(psn_seed + n));
			}

			float V_mem, n_channel, m_channel, h_channel;
			float v1, v2, v3, v4;
			float n1, n2, n3, n4;
			float m1, m2, m3, m4;
			float h1, h2, h3, h4;
			V_mem = V_m[n];
			n_channel = n_ch[n];
			m_channel = m_ch[n];
			h_channel = h_ch[n];
			v1 = hh_Vm(V_m[n], n_ch[n], m_ch[n], h_ch[n], I_syn_last[n], I_e[n], h);
			n1 = hh_n_ch(V_m[n], n_ch[n], h);
			m1 = hh_m_ch(V_m[n], m_ch[n], h);
			h1 = hh_h_ch(V_m[n], h_ch[n], h);
			V_m[n] = V_mem + v1/2.0f;
			n_ch[n] = n_channel + n1/2.0f;
			m_ch[n] = m_channel + m1/2.0f;
			h_ch[n] = h_channel + h1/2.0f;

			v2 = hh_Vm(V_m[n], n_ch[n], m_ch[n], h_ch[n], (I_syn[n] + I_psn[n] + I_syn_last[n])/2.0f, I_e[n], h);
			n2 = hh_n_ch(V_m[n], n_ch[n], h);
			m2 = hh_m_ch(V_m[n], m_ch[n], h);
			h2 = hh_h_ch(V_m[n], h_ch[n], h);
			V_m[n] = V_mem + v2/2.0f;
			n_ch[n] = n_channel + n2/2.0f;
			m_ch[n] = m_channel + m2/2.0f;
			h_ch[n] = h_channel + h2/2.0f;

			v3 = hh_Vm(V_m[n], n_ch[n], m_ch[n], h_ch[n], (I_syn[n] + I_psn[n] + I_syn_last[n])/2.0f, I_e[n], h);
			n3 = hh_n_ch(V_m[n], n_ch[n], h);
			m3 = hh_m_ch(V_m[n], m_ch[n], h);
			h3 = hh_h_ch(V_m[n], h_ch[n], h);
			V_m[n] = V_mem + v3;
			n_ch[n] = n_channel + n3;
			m_ch[n] = m_channel + m3;
			h_ch[n] = h_channel + h3;

			v4 = hh_Vm(V_m[n], n_ch[n], m_ch[n], h_ch[n], I_syn[n] + I_psn[n], I_e[n], h);
			n4 = hh_n_ch(V_m[n], n_ch[n], h);
			m4 = hh_m_ch(V_m[n], m_ch[n], h);
			h4 = hh_h_ch(V_m[n], h_ch[n], h);
			V_m[n] = V_mem + (v1 + 2.0f*v2 + 2.0f*v3 + v4)/6.0f;
			n_ch[n] = n_channel + (n1 + 2.0f*n2 + 2.0f*n3 + n4)/6.0f;
			m_ch[n] = m_channel + (m1 + 2.0f*m2 + 2.0f*m3 + m4)/6.0f;
			h_ch[n] = h_channel + (h1 + 2.0f*h2 + 2.0f*h3 + h4)/6.0f;

			// checking if there's spike on neuron
			if (V_m[n] > V_peak && V_mem > V_m[n] && V_m_last[n] <= V_mem){
				spike_time[Nneur*num_spike_neur[n] + n] = t;
				num_spike_neur[n]++;
			}
			V_m_last[n] = V_mem;
			I_syn_last[n] = I_syn[n] + I_psn[n];

//			if (n == 0){
//				printf("%.3f;%g;%g;%g;%g;%g;%g\n",
//						t*h, V_m[n], V_m[n+1], I_psn[n], I_psn[n+1], I_syn[n], I_syn[n+1]);
//			}
		}
}

using namespace std;

int main(int argc, char* argv[]){
	init_params(argc, argv);
	exp_psc = expf(-h/tau_psc);
	time_part_syn = 10.0f/h;
	T_sim = SimulationTime/h;
	init_neurs_from_file();
	init_conns_from_file();
	copy2device();
//	clear_files();
//	hipError_t er;

	init_poisson<<<dim3(Nneur/NEUR_BLOCK_SIZE + 1), dim3(NEUR_BLOCK_SIZE)>>>(psn_times_dev, psn_seeds_dev, seed, rate, h, Nneur, W_P_BUND_SZ);
	clock_t start = clock();
	time_t curr_time = time(0);
    char* st = asctime(localtime(&curr_time));
	cerr << "Start: " << st << endl;
    for (int t = 1; t < T_sim; t++){
		integrate_neurons<<<dim3(Nneur/NEUR_BLOCK_SIZE + 1), dim3(NEUR_BLOCK_SIZE)>>>(V_ms_dev, V_ms_last_dev, n_chs_dev, m_chs_dev, h_chs_dev, spike_times_dev, num_spikes_neur_dev,
				I_es_dev, ys_dev, I_syns_dev, y_psns_dev, I_psns_dev, psn_times_dev, psn_seeds_dev, I_last_dev, exp_w_p_dev, exp_psc, rate, Nneur, t, h);
		hipDeviceSynchronize();
		integrate_synapses<<<dim3(Ncon/SYN_BLOCK_SIZE + 1), dim3(SYN_BLOCK_SIZE)>>>(ys_dev, weights_dev, delays_dev, pre_conns_dev, post_conns_dev,
				spike_times_dev, num_spikes_syn_dev, num_spikes_neur_dev, t, Nneur, Ncon);
		hipDeviceSynchronize();
//		if(er != hipSuccess){
//			cerr << hipGetErrorString(er) << endl;
//		}

		if ((t % T_sim_partial) == 0){
			cout << t*h << endl;
			CUDA_CHECK_RETURN(hipMemcpy(spike_times, spike_times_dev, Nneur*sizeof(int)*T_sim_partial/time_part_syn, hipMemcpyDeviceToHost));
			CUDA_CHECK_RETURN(hipMemcpy(num_spikes_neur, num_spikes_neur_dev, Nneur*sizeof(int), hipMemcpyDeviceToHost));
			CUDA_CHECK_RETURN(hipMemcpy(num_spikes_syn, num_spikes_syn_dev, Ncon*sizeof(int), hipMemcpyDeviceToHost));
			swap_spikes();
			hipMemcpy(spike_times_dev, spike_times, Nneur*sizeof(int)*T_sim_partial/time_part_syn, hipMemcpyHostToDevice);
			hipMemcpy(num_spikes_neur_dev, num_spikes_neur, Nneur*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(num_spikes_syn_dev, num_spikes_syn, Ncon*sizeof(int), hipMemcpyHostToDevice);
		}
	}
	hipDeviceSynchronize();
	hipMemcpy(spike_times, spike_times_dev, Nneur*sizeof(int)*T_sim_partial/time_part_syn, hipMemcpyDeviceToHost);
	hipMemcpy(num_spikes_neur, num_spikes_neur_dev, Nneur*sizeof(int), hipMemcpyDeviceToHost);
	curr_time = time(0);
	cerr << "Stop: " << asctime(localtime(&curr_time)) << endl;
	cerr << "Finished!" << endl;

//	float s_time = ((float) clock() - (float) start)*1000./CLOCKS_PER_SEC;
//	cerr << "Elapsed time: " << s_time << " ms" << endl;
	save2HOST();
	save2file();
	return 0;
}

void init_conns_from_file(){
	int Ncon_part;

	ifstream con_file;
	con_file.open(par_f_name);
	con_file >> Ncon_part;
	Ncon = Ncon_part*W_P_NUM_BUND*NUM_BUND;
//	cerr << "Number of connections: " << Ncon << endl;
	malloc_conn_memory();
	float delay;
	int pre, post;

	for (int s = 0; s < Ncon_part; s++){
		con_file >> pre >> post >> delay;
		for (int bund = 0; bund < W_P_NUM_BUND*NUM_BUND; bund++){
			int idx = bund*Ncon_part + s;
			pre_conns[idx] = pre + bund*BUND_SZ;
			post_conns[idx] = post + bund*BUND_SZ;
			delays[idx] = delay/h;
			weights[idx] = (expf(1.0f)/tau_psc)*w_n;
		}
	}
	con_file.close();
}

void init_neurs_from_file(){
	unsigned int* param_distr_seeds = new unsigned int[Nneur]();
	for (int i = 0; i < Nneur; i++){
		int neur = i % BUND_SZ;
		param_distr_seeds[i] = 100000*(neur + 1);
		get_random(param_distr_seeds + i);
//		get_random(&param_distr_seeds[i]);
	}
	malloc_neur_memory();
	for (int bund = 0; bund < W_P_NUM_BUND; bund++){
		for (int n = 0; n < W_P_BUND_SZ; n++){
			int idx = W_P_BUND_SZ*bund + n;

			// IV on limit cycle
			V_ms[idx] = 32.9066f;
			V_ms_last[idx] = 32.9065f;
			n_chs[idx] = 0.574678f;
			m_chs[idx] = 0.913177f;
			h_chs[idx] = 0.223994f;

			// IV at equilibrium state
//			V_ms[idx] = -60.8457f;
//			V_ms_last[idx] = -60.8450f;
//			n_chs[idx] = 0.3763f;
//			m_chs[idx] = 0.0833f;
//			h_chs[idx] = 0.4636f;

//			unsigned int ivp_seed = seed + 1000 * n;
//			V_ms[idx] = -75.4989f + (32.9031f + 75.4989f) * get_random(&ivp_seed);
//			V_ms_last[idx] = V_ms[idx] - 0.001f;
//			n_chs[idx] = 0.3593f + (0.7574f - 0.3593f) * get_random(&ivp_seed);
//			m_chs[idx] = 0.0149f + (0.0149f - 0.9895f) * get_random(&ivp_seed);
//			h_chs[idx] = 0.0669f + (0.0669f - 0.5121f) * get_random(&ivp_seed);

//			I_es[idx] = I_e;
			float I_e_min = 5.22f;
			float I_e_max = 5.30f;
			I_es[idx] = I_e_min + (I_e_max-I_e_min)*get_random(param_distr_seeds + idx);
			exp_w_p[idx] = (expf(1.0f)/tau_psc)*(w_p_start + (w_p_stop - w_p_start)*bund/W_P_NUM_BUND);
		}
	}
}

void save2HOST(){
	int w_p_bund_idx, w_p_bund_neur, bund_idx, idx, neur;
	for (int n = 0; n < Nneur; n++){
		w_p_bund_idx = n/W_P_BUND_SZ;
		w_p_bund_neur = n - W_P_BUND_SZ*w_p_bund_idx;
		bund_idx = w_p_bund_neur/BUND_SZ;
		neur = w_p_bund_neur - BUND_SZ*bund_idx;
		idx = NUM_BUND*w_p_bund_idx + bund_idx;
		for (int sp_n = 0; sp_n < num_spikes_neur[n]; sp_n++){
			res_senders[W_P_NUM_BUND*NUM_BUND*num_spk_in_bund[idx] + idx] = neur;
			res_times[W_P_NUM_BUND*NUM_BUND*num_spk_in_bund[idx] + idx] = spike_times[Nneur*sp_n + n]*h;
			num_spk_in_bund[idx]++;
		}
	}
}

void swap_spikes(){
	int* spike_times_temp = new int[Nneur*T_sim_partial/time_part_syn];
	int* min_spike_nums_syn = new int[Nneur];
	for (int n = 0; n < Nneur; n++){
		min_spike_nums_syn[n] = INT_MAX;
	}
	for (int s = 0; s < Ncon; s++){
		if (num_spikes_syn[s] < min_spike_nums_syn[pre_conns[s]]){
			min_spike_nums_syn[pre_conns[s]] = num_spikes_syn[s];
		}
	}
	// В случае если у нейрона не было никаких исходящих связей, то минимальное количество
	// Спйков которые обрботли его исходящие синапсы будет равна INT_MAX, а это неверно
	// Поэтома надо насильно поставить 0, для этого тут и эта конструкция
	for (int n = 0; n < Nneur; n++){
		if (min_spike_nums_syn[n] == INT_MAX){
			min_spike_nums_syn[n] = 0;
		}
	}

	int w_p_bund_idx, w_p_bund_neur, bund_idx, neur, idx;
	for (int n = 0; n < Nneur; n++){
		w_p_bund_idx = n/W_P_BUND_SZ;
		w_p_bund_neur = n - W_P_BUND_SZ*w_p_bund_idx;
		bund_idx = w_p_bund_neur/BUND_SZ;
		neur = w_p_bund_neur - BUND_SZ*bund_idx;
		idx = NUM_BUND*w_p_bund_idx + bund_idx;
		for (int sp_n = 0; sp_n < min_spike_nums_syn[n]; sp_n++){
			res_senders[W_P_NUM_BUND*NUM_BUND*num_spk_in_bund[idx] + idx] = neur;
			res_times[W_P_NUM_BUND*NUM_BUND*num_spk_in_bund[idx] + idx] = spike_times[Nneur*sp_n + n]*h;
			num_spk_in_bund[idx]++;
		}

		for (int sp_n = min_spike_nums_syn[n]; sp_n < num_spikes_neur[n]; sp_n++){
			spike_times_temp[Nneur*(sp_n - min_spike_nums_syn[n]) + n] = spike_times[Nneur*sp_n + n];
		}
		num_spikes_neur[n] = num_spikes_neur[n] - min_spike_nums_syn[n];
	}

	for (int s = 0; s < Ncon; s++){
		num_spikes_syn[s] = num_spikes_syn[s] - min_spike_nums_syn[pre_conns[s]];
	}

	free(spike_times);
	free(min_spike_nums_syn);
	spike_times = spike_times_temp;
}

void save2file(){
	FILE* file;
	stringstream s;
	char* name = new char[500];
	for (int i = 0; i < W_P_NUM_BUND; i++){
		for (int j = 0; j < NUM_BUND; j++){
			s << f_name << "/"<< j << "/w_p_" << w_p_start + (w_p_stop - w_p_start)*i/W_P_NUM_BUND << endl;
			s >> name;
			file = fopen(name, "w");
			int idx = NUM_BUND*i + j;
			for (int spk = 0; spk < num_spk_in_bund[idx]; spk++){
				fprintf(file, "%.3f;%i\n", res_times[W_P_NUM_BUND*NUM_BUND*spk + idx], res_senders[W_P_NUM_BUND*NUM_BUND*spk + idx]);
			}
			fclose(file);
		}
	}

}

void malloc_neur_memory(){
	V_ms = new float[Nneur];
	V_ms_last = new float[Nneur];
	m_chs = new float[Nneur];
	n_chs = new float[Nneur];
	h_chs = new float[Nneur];
	I_es = new float[Nneur];

	ys = new float[Nneur]();
	I_syns = new float[Nneur]();
	y_psns = new float[Nneur]();
	I_psns = new float[Nneur]();

	I_last = new float[Nneur]();

	exp_w_p = new float[Nneur];

	// if num-th spike occur at a time t on n-th neuron then,
	// t is stored in element with index Nneur*num + n
	// spike_times[Nneur*num + n] = t
	spike_times = new int[Nneur*T_sim_partial/time_part_syn]();
	num_spikes_neur = new int[Nneur]();
	int expected_spk_num = BUND_SZ*SimulationTime/5.0f;
	res_times = new float[W_P_NUM_BUND*NUM_BUND*expected_spk_num];
	res_senders = new int[W_P_NUM_BUND*NUM_BUND*expected_spk_num];
	num_spk_in_bund = new int[W_P_NUM_BUND*NUM_BUND]();
}

void malloc_conn_memory(){
	weights = new float[Ncon];
	pre_conns = new int[Ncon];
	post_conns = new int[Ncon];
	delays = new int[Ncon];
	num_spikes_syn = new int[Ncon]();
}

void copy2device(){
	size_t n_fsize = Nneur*sizeof(float);
	size_t n_isize = Nneur*sizeof(int);
	size_t s_fsize = Ncon*sizeof(float);
	size_t s_isize = Ncon*sizeof(int);
	size_t spike_times_sz = n_isize*T_sim_partial/time_part_syn;

	// Allocating memory for array which contain var's for each neuron
	CUDA_CHECK_RETURN(hipMalloc((void**) &V_ms_dev, n_fsize));
	hipMalloc((void**) &V_ms_last_dev, n_fsize);
	hipMalloc((void**) &m_chs_dev, n_fsize);
	hipMalloc((void**) &n_chs_dev, n_fsize);
	hipMalloc((void**) &h_chs_dev, n_fsize);
	hipMalloc((void**) &I_es_dev, n_fsize);

	hipMalloc((void**) &ys_dev, n_fsize);
	hipMalloc((void**) &I_syns_dev, n_fsize);
	hipMalloc((void**) &y_psns_dev, n_fsize);
	hipMalloc((void**) &I_psns_dev, n_fsize);

	hipMalloc((void**) &I_last_dev, n_fsize);

	hipMalloc((void**) &exp_w_p_dev, n_fsize);
	hipMalloc((void**) &spike_times_dev, spike_times_sz);
	hipMalloc((void**) &num_spikes_neur_dev, n_isize);

	hipMalloc((void**) &psn_times_dev, n_isize);
	hipMalloc((void**) &psn_seeds_dev, n_isize);

	// Allocating memory for array which contain var's for each synapse
	hipMalloc((void**) &weights_dev, s_fsize);
	hipMalloc((void**) &pre_conns_dev, s_isize);
	hipMalloc((void**) &post_conns_dev, s_isize);
	hipMalloc((void**) &delays_dev, s_isize);
	hipMalloc((void**) &num_spikes_syn_dev, s_isize);

	// Copyng to GPU device memory neuron arrays
	hipMemcpy(V_ms_dev, V_ms, n_fsize, hipMemcpyHostToDevice);
	hipMemcpy(V_ms_last_dev, V_ms_last, n_fsize, hipMemcpyHostToDevice);
	hipMemcpy(m_chs_dev, m_chs, n_fsize, hipMemcpyHostToDevice);
	hipMemcpy(n_chs_dev, n_chs, n_fsize, hipMemcpyHostToDevice);
	hipMemcpy(h_chs_dev, h_chs, n_fsize, hipMemcpyHostToDevice);
	hipMemcpy(I_es_dev, I_es, n_fsize, hipMemcpyHostToDevice);

	hipMemcpy(ys_dev, ys, n_fsize, hipMemcpyHostToDevice);
	hipMemcpy(I_syns_dev, I_syns, n_fsize, hipMemcpyHostToDevice);
	hipMemcpy(I_psns_dev, I_psns, n_fsize, hipMemcpyHostToDevice);
	hipMemcpy(y_psns_dev, y_psns, n_fsize, hipMemcpyHostToDevice);

	hipMemcpy(I_last_dev, I_last, n_fsize, hipMemcpyHostToDevice);

	hipMemcpy(exp_w_p_dev, exp_w_p, n_fsize, hipMemcpyHostToDevice);

	hipMemcpy(spike_times_dev, spike_times, spike_times_sz, hipMemcpyHostToDevice);
	hipMemcpy(num_spikes_neur_dev, num_spikes_neur, n_isize, hipMemcpyHostToDevice);

	hipMemcpy(weights_dev, weights, s_fsize, hipMemcpyHostToDevice);
	hipMemcpy(pre_conns_dev, pre_conns, s_isize, hipMemcpyHostToDevice);
	hipMemcpy(post_conns_dev, post_conns, s_isize, hipMemcpyHostToDevice);
	hipMemcpy(delays_dev, delays, s_isize, hipMemcpyHostToDevice);
	hipMemcpy(num_spikes_syn_dev, num_spikes_syn, s_isize, hipMemcpyHostToDevice);
}

void clear_files(){
	FILE** files = new FILE*[W_P_NUM_BUND];
	stringstream s;
	char* name = new char[500];
	for (int i = 0; i < W_P_NUM_BUND; i++){
		s << f_name << "/w_p_" << w_p_start + (w_p_stop - w_p_start)*i/W_P_NUM_BUND << endl;
		s >> name;
		files[i] = fopen(name, "w");
		fclose(files[i]);
	}
}

void init_params(int argc, char* argv[]){
	stringstream str;
	for (int i = 1; i < argc; i++){
		str << argv[i] << endl;
		switch (i){
			case 1: str >> SimulationTime; break;
			case 2: str >> h; break;
			case 3: str >> Nneur; break;
			case 4: str >> W_P_NUM_BUND; break;
			case 5: str >> BUND_SZ; break;
			case 6: str >> f_name; break;
			case 7: str >> seed; break;
			case 8: str >> rate; break;
			case 9: str >> w_p_start; break;
			case 10: str >> w_p_stop; break;
			case 11: str >> w_n; break;
			case 12: str >> par_f_name; break;
		}
	}
	W_P_BUND_SZ = Nneur/W_P_NUM_BUND;
	NUM_BUND = W_P_BUND_SZ/BUND_SZ;
}
